
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 100
#define GRID_SIZE 100
#define N GRID_SIZE * BLOCK_SIZE

__global__ void VectorAdd (int *A, int *B, int *C) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	C[x] = A[x] + B[x];
}

int main () {
	int *hA, *hB, *hC;
	int *dA, *dB, *dC;
	int size = N * sizeof(int);
	int i;
	printf ("%d", size);
	// STEP 1 : Allocate memory for Host and Device variables
	hA = (int *) malloc(size);
	hB = (int *) malloc(size);
	hC = (int *) malloc(size);
	
	hipMalloc((void **)&dA, size);
	hipMalloc((void **)&dB, size);
	hipMalloc((void **)&dC, size);

	for (i = 0; i < N ; i++) {
		hA[i] = i;
		hB[i] = 2*i;
	}

	printf("\n Arrays to be added are:\n");
	printf("Array A:\n");
	for (i = 0; i < N ; i++) {
		printf("%d ", hA[i]);
	}

	printf("\nArray B:\n");
	for (i = 0; i < N ; i++) {
		printf("%d ", hB[i]);
	}

	// STEP 2: Copy data from Host to Device
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

	// STEP 3: Kernel Launch
	VectorAdd<<<GRID_SIZE, BLOCK_SIZE>>> (dA, dB, dC);

	// STEP 4: Copy results back to Host
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	// STEP 5 : Print the result
	printf("\n\nVector Addition is:\n");
	for (i = 0; i < N ; i++) {
		printf("%d ", hC[i]);
	}
	return 0;
}
